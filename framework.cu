#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "kernel.cu"
#include "kernel_CPU.C"

#define N 128
#define ITERS 10

void createRandomCells(int *cells, int n) {
	for (int i = 0; i < n*n*n; i++)
		if ((float)rand() / (float)RAND_MAX > 0.5)
			cells[i] = 1;
		else
			cells[i] = 0;
}

int main(int argc, char **argv){
	int *cells = NULL; 	// cells computed by CPU
	int *cellsGPU = NULL;	// CPU buffer for GPU results
	int *dCells = NULL;	// cells computed by GPU

	// parse command line
	int device = 0;
	if (argc == 2) 
		device = atoi(argv[1]);
	if (hipSetDevice(device) != hipSuccess){
		fprintf(stderr, "Cannot set CUDA device!\n");
		exit(1);
	}
	hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        printf("Using device %d: \"%s\"\n", device, deviceProp.name);

	// create events for timing
	hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

	// allocate and set host memory
	cells = (int*)malloc(N*N*N*sizeof(cells[0]));
	cellsGPU = (int*)malloc(N*N*N*sizeof(cells[0]));
	createRandomCells(cells, N);
 
	// allocate and set device memory
	if (hipMalloc((void**)&dCells, N*N*N*sizeof(dCells[0])) != hipSuccess) {
		fprintf(stderr, "Device memory allocation error!\n");
		goto cleanup;
	}
	hipMemcpy(dCells, cells, N*N*N*sizeof(dCells[0]), hipMemcpyHostToDevice);

	// solve on CPU
        printf("Solving on CPU...\n");
	hipEventRecord(start, 0);
	solveCPU(&cells, N, ITERS);
	hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float time;
        hipEventElapsedTime(&time, start, stop);
        printf("CPU performance: %f megacells/s\n",
                float(N*N*N)*float(ITERS)/time/1e3f);

	// dummy copy, just to awake GPU
        hipMemcpy(cellsGPU, dCells, N*N*N*sizeof(dCells[0]), hipMemcpyDeviceToHost);

	// solve on GPU
	printf("Solving on GPU...\n");
	hipEventRecord(start, 0);
	solveGPU(&dCells, N, ITERS);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
	printf("GPU performance: %f megacells/s\n",
                float(N*N*N)*float(ITERS)/time/1e3f);

	// check GPU results
	hipMemcpy(cellsGPU, dCells, N*N*N*sizeof(dCells[0]), hipMemcpyDeviceToHost);
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
			for (int k = 0; k < N; k++)
				if (cellsGPU[i*N*N + j*N + k] != cells[i*N*N + j*N + k]){
					printf("Error detected at [%i, %i, %i]: %i should be %i.\n", i, j, k, cellsGPU[i*N*N + j*N + k], cells[i*N*N + j*N + k]);
					goto cleanup; // exit after the first error
				}

	printf("Test OK.\n");

cleanup:
	hipEventDestroy(start);
        hipEventDestroy(stop);

	if (dCells) hipFree(dCells);

	if (cells) free(cells);
	if (cellsGPU) free(cellsGPU);

	return 0;
}

