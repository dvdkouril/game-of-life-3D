
#include <hip/hip_runtime.h>
// write your code into this file
#define BLOCK_SIZE 8

/*
  int ** cells         input cells grid
  int *  cellsOut      ouput cells grid - result of one iteration
  int    n             grid x/y/z dimension
 */
__global__ void solveIteration(int *cells, int *cellsOut, int n) {
  
  int i = blockIdx.x * blockDim.x + threadIdx.x; // global x coord
  int j = blockIdx.y * blockDim.y + threadIdx.y; // global y coord
  int k = blockIdx.z * blockDim.z + threadIdx.z; // global z coord

  int tx = threadIdx.x;  // block-local x coord
  int ty = threadIdx.y;  // block-local y coord
  int tz = threadIdx.z;  // block-local z coord
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int bz = blockIdx.z;
  // alocating memory with 1 cell border
  __shared__ int cellsBlock[BLOCK_SIZE + 2][BLOCK_SIZE + 2][BLOCK_SIZE + 2];

  // TODO copy stuff from global memory to shared memory
  cellsBlock[tx + 1][ty + 1][tz + 1] = cells[i*n*n + j*n + k];
  // border - this is starting to look pretty fucked up
  if (tx == 0)
    cellsBlock[0][ty][tz] = cells[(i-1)*n*n + j*n + k];
  if (ty == 0)
    cellsBlock[tx][0][tz] = cells[i*n*n + (j-1)*n + k];
  if (tz == 0)
    cellsBlock[tx][ty][0] = cells[i*n*n + j*n + (k - 1)];
  if (tx == BLOCK_SIZE - 1)
    cellsBlock[BLOCK_SIZE + 1][ty][tz] = cells[(i+1)*n*n + j*n + k];
  if (ty == BLOCK_SIZE - 1)
    cellsBlock[tx][BLOCK_SIZE + 1][tz] = cells[i*n*n + (j+1)*n + k];
  if (tz == BLOCK_SIZE - 1)
    cellsBlock[tx][ty][BLOCK_SIZE + 1] = cells[i*n*n + j*n + (k+1)];
  // corners
  if ((tx == 0) && (ty == 0))
    cellsBlock[0][0][tz] = cells[(i-1)*n*n + (j-1)*n + (k+1)];
  
  if ((tx == BLOCK_SIZE - 1) && (ty == 0))
    cellsBlock[BLOCK_SIZE + 1][0][tz] = cells[i*n*n + j*n + (k+1)];
  
  if ((tx == BLOCK_SIZE - 1 ) && (ty == BLOCK_SIZE - 1))
    cellsBlock[0][0][tz] = cells[i*n*n + j*n + (k+1)];
  
  if ((tx == 0) && (ty == BLOCK_SIZE - 1))
    cellsBlock[0][0][tz] = cells[i*n*n + j*n + (k+1)];
  
  __syncthreads();
  
  // TODO use stuff from shared memory when computing alive neighbours
  int alive = 0;
  for (int ii = max(tx - 1, 0); ii <= min(tx + 1, BLOCK_SIZE - 1); ii++) {
    for (int jj = max(ty - 1, 0); jj <= min(ty + 1, BLOCK_SIZE - 1); jj++) {
      for (int kk = max(tz - 1, 0); kk <= min(tz + 1, BLOCK_SIZE - 1); kk++) {
	alive += cellsBlock[ii][jj][kk];
      }
    }
  }
  alive -= cellsBlock[tx][ty][tz];

  if (alive < 4 || alive > 5) {
    cellsOut[i*n*n + j*n + k] = 0;
  } else if (alive == 5) {
    cellsOut[i*n*n + j*n + k] = 1;
  } else {
    cellsOut[i*n*n + j*n + k] = cells[i*n*n + j*n + k];
  }
  
}

// debug
void printGrid(int *cells, int n) {
  printf("\n");
  //for (int i = 0; i < n; i++) {
  int i = 0;
    for (int j = 0; j < n; j++) {
      for (int k = 0; k < n; k++) {
	printf("%d ", cells[i*n*n + j*n + k]);
      }
      printf("\n");
    }
    printf("\n\n");
    //}
}

/* 
   int** dCells       input/output parameter (as far as I understand)
   int   n            grid x/y/z dimension
   int   iters        how many iteration are meant to be simulated
 */
void solveGPU(int **dCells, int n, int iters){
  // alocate array for computing next iteration
  int *cellsNextIter = NULL;
  // memory allocated on the graphics card (can't be accessed in this function!!!)
  if (hipMalloc((void**)&cellsNextIter, n*n*n*sizeof(cellsNextIter[0])) != hipSuccess) {
    printf("Device memory allocation error\n");
  }

  // debug
  //printf("first slide of input");
  //int *cellsToPrint = (int*)malloc(n*n*n*sizeof(int));
  //cudaMemcpy(cellsToPrint, cellsNextIter, n*n*n*sizeof(int), cudaMemcpyDeviceToHost);
  //cudaMemcpy(cellsToPrint, *dCells, n*n*n*sizeof(int), cudaMemcpyDeviceToHost);
  //printGrid(cellsToPrint, n);
  
  for (int i = 0; i < iters; i++) {
    // grid and block dimensions setup
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
    //dim3 dimGrid(n / BLOCK_SIZE, n / BLOCK_SIZE, n / BLOCK_SIZE);
    int blocksNum = (int)ceil(n / (float)(BLOCK_SIZE - 2));
    dim3 dimGrid(blocksNum, blocksNum, blocksNum);
    
    // kernel invocation
    solveIteration<<<dimGrid, dimBlock>>>(*dCells, cellsNextIter, n);

    // kernel invocation error checking
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));

    // swap grids
    int *tmp = *dCells;
    *dCells = cellsNextIter; // setting newly computed iteration to as the result
    cellsNextIter = tmp; // unnecessary

  }

  // debug
  //printf("first slide of output");
  //int *cellsToPrint = (int*)malloc(n*n*n*sizeof(int));
  //cudaMemcpy(cellsToPrint, *dCells, n*n*n*sizeof(int), cudaMemcpyDeviceToHost);
  //printGrid(cellsToPrint, n);

  // TODO free allocated memory
  hipFree(*dCells); // the memory that I allocated should end up in dCells
	
}

