
#include <hip/hip_runtime.h>
// write your code into this file
#define BLOCK_SIZE 8

/*
  int ** cells         input cells grid
  int *  cellsOut      ouput cells grid - result of one iteration
  int    n             grid x/y/z dimension
 */
__global__ void solveIteration(int *cells, int *cellsOut, int n) {
  
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;

  //printf("[%d, %d, %d] %d\n", i, j, k, i*n*n + j*n + k);
  //printf("&cells = %p, &cellsOut = %p, &n = %p \n", cells, cellsOut, &n);
  //cellsOut[i*n*n + j*n + k] = cells[i*n*n + j*n + k];
  //printf("thread [%d, %d, %d], n = %d\n", i, j, k, n);

  // searching the neighbourhood for alive cells
  int alive = 0; // number of alive neighbours
  for (int ii = max(i-1, 0); ii <= min(i+1, n-1); ii++)
    for (int jj = max(j-1, 0); jj <= min(j+1, n-1); jj++)
      for (int kk = max(k-1, 0); kk <= min(k+1, n-1); kk++)
	alive += cells[ii*n*n + jj*n + kk];
  alive -= cells[i*n*n + j*n + k];
  //printf("index %d, alive %d", i*n*n + j*n + k, alive);

  //cellsOut[i*n*n + j*n + k] = (*cells)[i*n*n + j*n + k];
  
  //cellsOut[i*n*n + j*n +k] = 0;
  /*cellsOut[0] = 0;
  cellsOut[1] = 1;
  cellsOut[2] = 2;
  cellsOut[3] = 3;
  cellsOut[4] = 4;
  cellsOut[5] = 5;*/
  /*for (int num = 0; num < n*n*n; num++) {
    cellsOut[num] = num;
    }*/
  //int current = (*cells)[i*n*n + j*n + k]; // debug
  //int result = 0; // debug
  int index = i*n*n + j*n +k;
  //printf("index %d, alive %d", index, alive);
  if (alive < 4 || alive > 5) {
    cellsOut[index] = 0;
  } else if (alive == 5) {
    cellsOut[index] = 1;
  } else {
    cellsOut[i*n*n + j*n + k] = cells[i*n*n + j*n + k];
  }
  // printf("[%d, %d, %d] from %d to %d", i, j, k, current, result);
  
}

// debug
void printGrid(int *cells, int n) {
  printf("\n");
  //for (int i = 0; i < n; i++) {
  int i = 0;
    for (int j = 0; j < n; j++) {
      for (int k = 0; k < n; k++) {
	printf("%d ", cells[i*n*n + j*n + k]);
      }
      printf("\n");
    }
    printf("\n\n");
    //}
}

/* 
   int** dCells       input/output parameter (as far as I understand)
   int   n            grid x/y/z dimension
   int   iters        how many iteration are meant to be simulated
 */
void solveGPU(int **dCells, int n, int iters){
  // alocate array for computing next iteration
  int *cellsNextIter = NULL;
  // memory allocated on the graphics card (can't be accessed in this function!!!)
  if (hipMalloc((void**)&cellsNextIter, n*n*n*sizeof(cellsNextIter[0])) != hipSuccess) {
    printf("Device memory allocation error\n");
  }
  //cudaMemset(cellsNextIter, 2, n*n*n*sizeof(cellsNextIter[0]));
  //cudaMemcpy(cellsNextIter, *dCells, n*n*n*sizeof(cellsNextIter[0]), cudaMemcpyDeviceToDevice);

  // debug
  //printf("first slide of input");
  //int *cellsToPrint = (int*)malloc(n*n*n*sizeof(int));
  //cudaMemcpy(cellsToPrint, cellsNextIter, n*n*n*sizeof(int), cudaMemcpyDeviceToHost);
  //cudaMemcpy(cellsToPrint, *dCells, n*n*n*sizeof(int), cudaMemcpyDeviceToHost);
  //printGrid(cellsToPrint, n);
  
  //int iterNum = 0;
  for (int i = 0; i < iters; i++) {
    // grid and block dimensions setup
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
    int threadsPerBlock = BLOCK_SIZE*BLOCK_SIZE*BLOCK_SIZE; 
    //dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(n / BLOCK_SIZE, n / BLOCK_SIZE, n / BLOCK_SIZE);
    // kernel invocation
    solveIteration<<<dimGrid, dimBlock>>>(*dCells, cellsNextIter, n);
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));

    // swap grids
    int *tmp = *dCells;
    *dCells = cellsNextIter; // setting newly computed iteration to as the result
    cellsNextIter = tmp; // unnecessary

    //iterNum = i; // debug
  }

  // debug
  //printf("first slide of output");
  //int *cellsToPrint = (int*)malloc(n*n*n*sizeof(int));
  //cudaMemcpy(cellsToPrint, *dCells, n*n*n*sizeof(int), cudaMemcpyDeviceToHost);
  //printGrid(cellsToPrint, n);
  
  //printf("number of iteration executed: %d", iterNum);

  // TODO free allocated memory
	
}

